#include "hip/hip_runtime.h"
#include "plonky2_gpu_impl.cuh"

#include <vector>
#include <fstream>
#undef CUDA_ASSERT

#define CUDA_ASSERT(expr) \
do {\
    if (auto code = expr; code != hipSuccess) {\
        printf("%s@%d failed: %s\n", #expr, __LINE__, hipGetErrorString(code));\
        return -1;\
    }\
} while(0)

#include <chrono>
#include <iostream>

static inline int ceil(int v, int v2) {
    assert(v < v2);
    return (v2+v-1)/v * v;
}
__global__
void test()
{
//    GoldilocksField n_inv = {.data = 18446673700670423041ULL};
//    GoldilocksField v = {.data = 0x000000002EF7A1BC};
//    printf("inv: %016lX\n", n_inv.data);
//    printf("n  : %016lX\n", v.data);
//    printf("res: %016lX\n", (n_inv * v - n_inv).data);

    GoldilocksField data[8] = {
	{12057761340118092379ULL},
	{6921394802928742357ULL},
	{401572749463996457ULL},
	{8075242603528285606ULL},
	{16383556155787439553ULL},
	{18045582516498195573ULL},
	{7296969412159674050ULL},
	{8317318176954617326ULL}
    };

    GoldilocksField state[SPONGE_WIDTH] = {0};
    for (int k = 0; k < SPONGE_RATE; ++k)
        state[k] = data[k];
    PoseidonHasher::permute_poseidon(state);
    auto out =  *(PoseidonHasher::HashOut*)state;
    PRINT_HEX("hash", out);
}

template <class T>
std::vector<T> read_vec_from_bin(std::string filename) {
    std::vector<T> res;
    std::ifstream file(filename, std::ios::binary);

    // 获取文件大小
    file.seekg(0, std::ios::end);
    std::streampos fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    // 根据文件大小调整vector容量
    res.resize(fileSize / sizeof(T));

    // 从文件中读取数据到vector
    file.read(reinterpret_cast<char*>(res.data()), fileSize);
    return res;
};

//auto read_hvec_from_bin = [](std::string filename) -> std::vector<PoseidonHasher::HashOut>{
//    std::vector<PoseidonHasher::HashOut> res;
//    std::ifstream file(filename, std::ios::binary);
//
//    // 获取文件大小
//    file.seekg(0, std::ios::end);
//    std::streampos fileSize = file.tellg();
//    file.seekg(0, std::ios::beg);
//
//    // 根据文件大小调整vector容量
//    res.resize(fileSize / sizeof(PoseidonHasher::HashOut));
//
//    // 从文件中读取数据到vector
//    file.read(reinterpret_cast<char*>(res.data()), fileSize);
//    return res;
//}

template <class T>
struct DataSlice{
    T* ptr;
    int len;
};

int main()
{
    hipStream_t stream;
    hipSetDevice(0);
    hipDeviceReset();
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
//    test<<<1, 1, 0, stream>>>();
//    hipStreamSynchronize(stream);
//
//    exit(0);

    int poly_num = 234, values_num_per_poly = 262144, log_len = 18;
    constexpr int rate_bits = 3;
    int values_num_per_extpoly = values_num_per_poly*(1<<rate_bits);
    int cap_height = 4;
    int len_cap = 1 << cap_height;
    int salt_size = 0;
    int num_digests = 2 * (values_num_per_extpoly - len_cap);
    int num_digests_and_caps = num_digests + len_cap;
    int thcnt = 0;
    int nthreads = 32;
    int ext_poly_num = poly_num + salt_size;

    double ifft_kernel_use, lde_kernel_use, mul_shift_kernel_use, fft_kernel_use, reverse_index_bits_kernel_use,
                hash_leaves_kernel_use, reduce_digests_kernel_use, transpose_kernel_use;

#define  read_fvec_from_bin read_vec_from_bin<GoldilocksField>
#define  read_hvec_from_bin read_vec_from_bin<PoseidonHasher::HashOut>

    auto values_flatten = read_fvec_from_bin("values.bin");
    auto root_table  = read_fvec_from_bin("roots.bin");
    auto root_table2 = read_fvec_from_bin("roots2.bin");
    auto shift_powers = read_fvec_from_bin("powers.bin");


//    auto originalVector = values_flatten;
//    values_flatten.insert(values_flatten.end(), originalVector.begin(), originalVector.end());
//    poly_num *= 2;

    GoldilocksField *d_values_flatten;
    CUDA_ASSERT(hipMalloc(&d_values_flatten, values_num_per_poly*poly_num*sizeof(GoldilocksField)));

    CUDA_ASSERT(hipMemcpyAsync(d_values_flatten, &values_flatten[0],  values_num_per_poly*poly_num*sizeof(GoldilocksField),
                                hipMemcpyHostToDevice, stream));
    hipStreamSynchronize(stream);

    int pad_extvalues_len = values_num_per_extpoly*ext_poly_num;
    GoldilocksField *d_ext_values_flatten;
    CUDA_ASSERT(hipMalloc(&d_ext_values_flatten,
                            (pad_extvalues_len + values_num_per_poly*ext_poly_num*(1<<rate_bits) + num_digests_and_caps*4)*sizeof(GoldilocksField)));
    d_ext_values_flatten += pad_extvalues_len;

    GoldilocksField *d_root_table;
    CUDA_ASSERT(hipMalloc(&d_root_table, (values_num_per_poly+1)*sizeof(GoldilocksField)));

    CUDA_ASSERT(hipMemcpyAsync(d_root_table, &root_table[0],  (values_num_per_poly+1)*sizeof(GoldilocksField),
                                hipMemcpyHostToDevice, stream));

    hipStreamSynchronize(stream);

    GoldilocksField *d_root_table2;
    CUDA_ASSERT(hipMalloc(&d_root_table2, (values_num_per_poly*(1<<rate_bits)+1) * sizeof(GoldilocksField)));

    CUDA_ASSERT(hipMemcpyAsync(d_root_table2, &root_table2[0],  (values_num_per_poly*(1<<rate_bits)+1) * sizeof(GoldilocksField),
                                hipMemcpyHostToDevice, stream));

    hipStreamSynchronize(stream);

    GoldilocksField *d_shift_powers;
    CUDA_ASSERT(hipMalloc(&d_shift_powers, values_num_per_poly * sizeof(GoldilocksField)));

    CUDA_ASSERT(hipMemcpyAsync(d_shift_powers, &shift_powers[0],  values_num_per_poly * sizeof(GoldilocksField),
                                hipMemcpyHostToDevice, stream));

    hipStreamSynchronize(stream);

//    printf("buf0: ");
//    for (int i = (1<<20); i < 8+(1<<20); ++i) {
//        printf("%016lX, ", values_flatten[i].data);
//    }
//    printf("\n");

//    CudaInvContext ctx = {.stream = stream};
    GoldilocksField n_inv = {.data = 18446673700670423041ULL};

//    ifft(d_values_flatten, poly_num, values_num_per_poly, log_len, d_root_table, &n_inv, &ctx);

//    reverse_index_bits_kernel<<<poly_num, 32, 0, stream>>>(d_values_flatten, poly_num, values_num_per_poly, log_len);
//    hipStreamSynchronize(stream);
//
//    {
//        CUDA_ASSERT(hipMemcpyAsync(&values_flatten[0], d_values_flatten,  values_num_per_poly*poly_num*sizeof(GoldilocksField),
//                                    hipMemcpyDeviceToHost, stream));
//        hipStreamSynchronize(stream);
//
//        std::ofstream file("res-gpu-bits.bin", std::ios::binary);
//        if (file.is_open()) {
//            file.write(reinterpret_cast<const char*>(values_flatten.data()), values_flatten.size() * sizeof(uint64_t));
//            file.close();
//            std::cout << "Data written to file." << std::endl;
//        } else {
//            std::cerr << "Failed to open file." << std::endl;
//        }
//
//    }





    clock_t start = clock();
//        hipMemsetAsync(d_ext_values_flatten, 8*values_num_per_poly*poly_num*(1<<rate_bits), 0, ctx->stream2);
    ifft_kernel<<<poly_num, 32*8, 0, stream>>>(d_values_flatten, poly_num, values_num_per_poly, log_len, d_root_table, n_inv);
    hipStreamSynchronize(stream);
//        hipStreamSynchronize(ctx->stream2);
    printf("ifft_kernel elapsed: %.2lf\n", ifft_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

    start = clock();
    thcnt = values_num_per_poly*poly_num;
    nthreads = 32;
    lde_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_values_flatten, d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits);
    hipStreamSynchronize(stream);
    printf("lde_kernel elapsed: %.2lf\n", lde_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

    start = clock();
    thcnt = values_num_per_poly*poly_num;
    nthreads = 32;
    init_lde_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits);
    hipStreamSynchronize(stream);
    printf("init_lde_kernel elapsed: %.2lf\n", (double )(clock()-start) / CLOCKS_PER_SEC * 1000);

    start = clock();
    thcnt = values_num_per_poly*poly_num;
    nthreads = 32;
    mul_shift_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly, rate_bits, d_shift_powers);
    hipStreamSynchronize(stream);
    printf("mul_shift_kernel elapsed: %.2lf\n", mul_shift_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

    start = clock();
    fft_kernel<<<poly_num, 32*8, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_poly*(1<<rate_bits), log_len+rate_bits, d_root_table2, rate_bits);
    hipStreamSynchronize(stream);
    printf("fft_kernel elapsed: %.2lf\n", fft_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);


//    CUDA_ASSERT(hipMemcpyAsync(&values_flatten[0], d_values_flatten,  values_num_per_poly*poly_num*sizeof(GoldilocksField),
//                                hipMemcpyDeviceToHost, stream));
//    hipStreamSynchronize(stream);
//
//    std::ofstream file("values_flatten-gpu.bin", std::ios::binary);
//    if (file.is_open()) {
//        file.write(reinterpret_cast<const char*>(values_flatten.data()), values_flatten.size() * sizeof(uint64_t));
//        file.close();
//        std::cout << "Data written to file." << std::endl;
//    } else {
//        std::cerr << "Failed to open file." << std::endl;
//    }


    start = clock();
    thcnt = values_num_per_extpoly*poly_num;
    nthreads = 32;
    reverse_index_bits_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, poly_num, values_num_per_extpoly, log_len+rate_bits);
    hipStreamSynchronize(stream);
    printf("reverse_index_bits_kernel elapsed: %.2lf\n", reverse_index_bits_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

    int log2_leaves_len = log_len + rate_bits;
    assert(cap_height <= log2_leaves_len);

    auto *d_digest_buf = (PoseidonHasher::HashOut*)(d_ext_values_flatten + values_num_per_extpoly * ext_poly_num);

    start = clock();
    thcnt = values_num_per_extpoly;
    nthreads = 32;
    hash_leaves_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(
            d_ext_values_flatten, poly_num+salt_size, values_num_per_extpoly, d_digest_buf, len_cap, num_digests);
    hipStreamSynchronize(stream);
    printf("hash_leaves_kernel elapsed: %.2lf\n", hash_leaves_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

    start = clock();
    nthreads = 32*8;
    thcnt = len_cap * nthreads;
    reduce_digests_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(values_num_per_extpoly, d_digest_buf, len_cap, num_digests);
    hipStreamSynchronize(stream);
    printf("reduce_digests_kernel elapsed: %.2lf\n", reduce_digests_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

//    printf("error: %s\n", hipGetErrorString(hipGetLastError()));

//    std::vector<PoseidonHasher::HashOut> outs(num_digests_and_caps);
//    CUDA_ASSERT(hipMemcpyAsync(&outs[0], d_digest_buf,  num_digests_and_caps*sizeof(PoseidonHasher::HashOut), hipMemcpyDeviceToHost, stream));
//    hipStreamSynchronize(stream);
////    for (int i = 0; i < outs.size(); ++i) {
////        printf("idx: %d, ", i);
////        PRINT_HEX("hash", outs[i]);
////    }
//
//    PoseidonHasher::HashOut* cap_buf = &outs[num_digests];
//
//    for (int i = 0; i < len_cap; ++i) {
//        printf("cap idx: %d, ", i);
//        PRINT_HEX("hash", cap_buf[i]);
//    }

    start = clock();
    thcnt = values_num_per_extpoly;
    nthreads = 32;
    transpose_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(d_ext_values_flatten, d_ext_values_flatten - pad_extvalues_len, ext_poly_num, values_num_per_extpoly);
    hipStreamSynchronize(stream);
    printf("transpose_kernel elapsed: %.2lf\n", transpose_kernel_use=(double )(clock()-start) / CLOCKS_PER_SEC * 1000);

//    start = clock();
//    for (int i = 0 ; i < 100; ++i) {
//        CUDA_ASSERT(hipMemcpyAsync(&values_flatten[0 + i*2048], d_values_flatten + i *2048*100,  2048, hipMemcpyDeviceToHost, stream));
//        hipStreamSynchronize(stream);
//    }
//    printf("test: %.2lf\n", (double )(clock()-start) / CLOCKS_PER_SEC * 1000);

    double total_use =
            ifft_kernel_use+
            lde_kernel_use+
            mul_shift_kernel_use+
            fft_kernel_use+
            reverse_index_bits_kernel_use+
            hash_leaves_kernel_use+
            reduce_digests_kernel_use+
            transpose_kernel_use;

    printf("total use:%.2lf\n", total_use);

    uint8_t *start_p = (uint8_t*)d_ext_values_flatten;
    uint8_t *end_p   = (uint8_t*)(d_ext_values_flatten+values_num_per_extpoly*ext_poly_num);
//    {
//        std::vector<GoldilocksField> data;
//        uint8_t *old_p;
//        DataSlice<std::remove_reference<decltype(data[0])>::type>{(decltype(&data[0])) old_p, (int)data.size()};
//    }
//    std::remove_reference<decltype(data2[0])>::type asdf;

#define DO_MEMCPY_TODEV(data) \
            int cpylen = data.size() * sizeof(data[0]);                  \
            CUDA_ASSERT(hipMemcpyAsync(start_p, &data[0],  cpylen, hipMemcpyHostToDevice, stream)); \
            uint8_t *old_p = start_p; \
            start_p += cpylen;\
            assert(start_p < end_p);                  \
            DataSlice<std::remove_reference<decltype(data[0])>::type>{(decltype(&data[0])) old_p, (int)data.size()};

#define  read_fvec_to_dev(fname) \
    ({                             \
        auto data = read_vec_from_bin<GoldilocksField>(fname); \
        DO_MEMCPY_TODEV(data)                             \
    })

#define  read_hvec_to_dev(fname) \
    ({                             \
        auto data = read_vec_from_bin<PoseidonHasher::HashOut>(fname); \
        DO_MEMCPY_TODEV(data)                             \
    })

    auto zs_partial_products_commitment_polynomials = read_fvec_to_dev("zs_partial_products_commitment.polynomials.bin");
    auto zs_partial_products_commitment_leaves      = read_fvec_to_dev("zs_partial_products_commitment.leaves.bin");
    auto zs_partial_products_commitment_digests     = read_hvec_to_dev("zs_partial_products_commitment.digests.bin");
    auto zs_partial_products_commitment_caps        = read_hvec_to_dev("zs_partial_products_commitment.caps.bin");

    auto constants_sigmas_commitment_polynomials    = read_fvec_to_dev("constants_sigmas_commitment.polynomials.bin");
    auto constants_sigmas_commitment_leaves         = read_fvec_to_dev("constants_sigmas_commitment.leaves.bin");
    auto constants_sigmas_commitment_digests        = read_hvec_to_dev("constants_sigmas_commitment.digests.bin");
    auto constants_sigmas_commitment_caps           = read_hvec_to_dev("constants_sigmas_commitment.caps.bin");

    auto k_is = read_fvec_to_dev("k_is.bin");
    auto alphas = read_fvec_to_dev("alphas.bin");
    auto betas = read_fvec_to_dev("betas.bin");
    auto gammas = read_fvec_to_dev("gammas.bin");
    auto points = read_fvec_to_dev("points.bin");
    auto z_h_on_coset_evals = read_fvec_to_dev("z_h_on_coset.evals.bin");
    auto z_h_on_coset_inverses = read_fvec_to_dev("z_h_on_coset.inverses.bin");

    GoldilocksField *d_outs;
    CUDA_ASSERT(hipMalloc(&d_outs, values_num_per_extpoly*2*sizeof(GoldilocksField)));

    hipStreamSynchronize(stream);
    size_t total_dev_use = start_p-(uint8_t*)d_ext_values_flatten;
    printf("total_dev_use: %fG\n", (double )total_dev_use/1024/1024/1024);

    int num_challenges = 2;
    int num_gate_constraints = 231;
    int num_constants = 8;
    int num_routed_wires = 80;
    int quotient_degree_factor = 8;
    int num_partial_products = 9;
    int constants_sigmas_commitment_leaf_len = 88;
    int zs_partial_products_commitment_leaf_len = 20;
    int wires_commitment_leaf_len = 234;

//    printf("%d, %d\n", constants_sigmas_commitment_leaves.len, values_num_per_extpoly*constants_sigmas_commitment_leaf_len);
    assert(constants_sigmas_commitment_leaves.len    == values_num_per_extpoly*constants_sigmas_commitment_leaf_len);
    assert(zs_partial_products_commitment_leaves.len == values_num_per_extpoly*zs_partial_products_commitment_leaf_len);
    assert(points.len == values_num_per_extpoly);
    assert(alphas.len == num_challenges);
    assert(betas.len == num_challenges);
    assert(gammas.len == num_challenges);

    start = clock();
    thcnt = 15000;
    nthreads = 32;
    printf("values_num_per_extpoly: %d, log_len: %d\n", values_num_per_extpoly, log_len);
    PoseidonHasher::HashOut public_inputs_hash = {
            GoldilocksField{0x672c5e6c12ad3476}, GoldilocksField{0xca5c2e49acfad27e},
            GoldilocksField{0x296be18388d15f70}, GoldilocksField{0x66b42e146a70d96d}
    };;
    compute_quotient_values_kernel<<<(thcnt+nthreads-1)/nthreads, nthreads, 0, stream>>>(
            log_len, rate_bits,
            points.ptr,
            d_outs,
            public_inputs_hash,

            constants_sigmas_commitment_leaves.ptr,     constants_sigmas_commitment_leaf_len,
            zs_partial_products_commitment_leaves.ptr,  zs_partial_products_commitment_leaf_len,
            d_ext_values_flatten - pad_extvalues_len,                wires_commitment_leaf_len,
            num_constants, num_routed_wires,
            num_challenges,
            num_gate_constraints,

            quotient_degree_factor,
            num_partial_products,

            z_h_on_coset_evals.ptr,
            z_h_on_coset_inverses.ptr,

            k_is.ptr,
            alphas.ptr,
            betas.ptr,
            gammas.ptr
    );
    hipStreamSynchronize(stream);
    printf("compute_quotient_values_kernel elapsed: %.2lf\n", (double )(clock()-start) / CLOCKS_PER_SEC * 1000);

    return 0;
}
